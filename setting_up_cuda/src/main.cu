#include "hip/hip_runtime.h"
/*
** EPITECH PROJECT, 2025
** File description:
** main
*/

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include "open_file.h"
#include "map_parser.h"
#include "square_utils.h"

__device__ int min3(int a, int b, int c)
{
    int m = (a < b) ? a : b;
    return (m < c) ? m : c;
}

// Each thread processes one element of the anti-diagonal "diag".
// The grid (dp) is of size width x height and stored in linear memory.
__global__ void dp_kernel(int *dp, int width, int height, int diag)
{
    // Calculate the thread index within the anti-diagonal.
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // For the anti-diagonal with index "diag", the column indices vary from "start" to "end".
    int start = diag - (height - 1);
    if (start < 0)
        start = 0;
    int end = diag;
    if (end >= width)
        end = width - 1;
    int nbElem = end - start + 1;

    if (index < nbElem) {
        int i = start + index;   // Column index.
        int j = diag - i;        // Row index.

        // Only process cells with i > 0 and j > 0 (borders remain unchanged).
        if (i > 0 && j > 0 && i < width && j < height) {
            // If the cell is empty (represented by 1), update its value;
            // if it is an obstacle (represented by 0), do nothing.
            if (dp[j * width + i] != 0) {
                int top      = dp[(j - 1) * width + i];
                int left     = dp[j * width + (i - 1)];
                int top_left = dp[(j - 1) * width + (i - 1)];
                dp[j * width + i] = min3(top, left, top_left) + 1;
            }
        }
    }
}

// Process the anti-diagonals on the GPU.
void process_dp_on_gpu(int *d_grid, int width, int height)
{
    int threadsPerBlock = 256;
    // The total number of anti-diagonals is (width + height - 1).
    for (int diag = 1; diag < width + height - 1; diag++) {
        int start = diag - (height - 1);
        if (start < 0)
            start = 0;
        int end = diag;
        if (end >= width)
            end = width - 1;
        int nbElem = end - start + 1;
        int blocks = (nbElem + threadsPerBlock - 1) / threadsPerBlock;

        dp_kernel<<<blocks, threadsPerBlock>>>(d_grid, width, height, diag);
        hipDeviceSynchronize(); // Ensure dependencies are respected.
    }
}

int main(int ac, const char* av[])
{
    if (ac != 2)
        return 84;

    char *map = open_file(av[1]);
    if (!map)
        return 84;
    
    int width, height;
    if (!get_map_dimensions(map, &width, &height)) {
        free(map);
        return 84;
    }
#ifdef DEBUG
    printf("Map size: %d x %d\n", width, height);
#endif

    int *grid = convert_map_to_grid(map, width, height);
    if (!grid) {
        free(map);
        return 84;
    }

    // Allocate the grid on the GPU (the dp array).
    int *d_grid;
    size_t grid_size = width * height * sizeof(int);
    hipError_t err = hipMalloc((void**)&d_grid, grid_size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc error: %s\n", hipGetErrorString(err));
        free(grid);
        free(map);
        return 84;
    }
    hipMemcpy(d_grid, grid, grid_size, hipMemcpyHostToDevice);

    process_dp_on_gpu(d_grid, width, height);

    // Retrieve the computed dp matrix from the GPU.
    int *dp_host = (int *)malloc(grid_size);
    if (!dp_host) {
        fprintf(stderr, "Error: malloc failed.\n");
        hipFree(d_grid);
        free(grid);
        free(map);
        return 84;
    }
    hipMemcpy(dp_host, d_grid, grid_size, hipMemcpyDeviceToHost);

    int max_size = 0, max_i = 0, max_j = 0;
    find_largest_square(dp_host, width, height, &max_size, &max_i, &max_j);
#ifdef DEBUG
    printf("Largest square size: %d at position (%d, %d)\n", max_size, max_i, max_j);
#endif


    mark_largest_square(map, width, height, max_size, max_i, max_j);

    print_final_map(map, width, height);

    free(dp_host);
    free(grid);
    hipFree(d_grid);
    free(map);

    return 0;
}
